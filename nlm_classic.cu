#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32

__device__ float Max(float x, float y)
{
    return (x > y) ? x : y;
}

__device__ float Min(float x, float y)
{
    return (x < y) ? x : y;
}

__device__ int Max(int x, int y)
{
    return (x > y) ? x : y;
}

__device__ int Min(int x, int y)
{
    return (x < y) ? x : y;
}

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__global__ void nlm_classic_global(const float* d_src,
                                   float* d_dst,
                                   int patch, int window,
                                   int width, int height, float fSigma2, float fH2, float icwl) {

    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if (ix < width && iy < height)
    {
        int i1 = ix+patch;
        int j1 = iy+patch;

        float wmax = 0;
        float average = 0;
        float sweight = 0;

        int rmin = Max(i1-window,patch+1);
        int rmax = Min(i1+window,width+patch);
        int smin = Max(j1-window,patch+1);
        int smax = Min(j1+window,height+patch);

        for (int r = rmin; r < rmax; r++) {
            for (int s = smin; s < smax; s++) {
                if (r == i1 && s == j1) {
                    continue;
                }
                float diff = 0;
                for (int ii = -patch; ii <= patch; ii++) {
                    for (int jj = -patch; jj <= patch; jj++) {
                        float a = d_src[width*(j1+jj)+(i1+ii)];
                        float b = d_src[width*(s+jj)+(r+ii)];
                        float c = a-b;
                        diff += c*c;
                    }
                }
                diff = Max(float(diff - 2.0 * (double) icwl *  fSigma2), 0.0f);
                diff = diff / fH2;
                float W = __expf(-diff);

                if (W > wmax) {
                    wmax = W;
                }

                sweight += W;
                average += W * d_src[width*s + r];
            }
        }
        average += wmax * d_src[width*j1+i1];
        sweight += wmax;

        if (sweight > 0) {
            d_dst[width*j1+i1] = average / sweight;
        }
        else {
            d_dst[width*j1+i1] = d_src[width*j1+i1];
        }
    }
}

void nlm_filter_classic_CUDA(const float* h_src, float* h_dst, int width, int height, float fSigma, float fParam, int patch, int window) {
    hipError_t err = hipSuccess;

    float* d_src = NULL, *d_dst = NULL;
    unsigned int nBytes = sizeof(float) * (width*height);

    err = hipMalloc((void **)& d_src, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector SRC (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)& d_dst, nBytes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector DST (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_src, h_src, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector SRC from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_dst, h_dst, nBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));

    int patchSize = patch*2+1;
    float fSigma2 = fSigma * fSigma;
    float fH = fParam * fSigma;
    float fH2 = fH * fH;
    float icwl = patchSize * patchSize;
    fH2 *= icwl;

    nlm_classic_global<<<grid, threads>>>(d_src, d_dst, patch, window, width, height, fSigma2, fH2, icwl);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch nlm_classic_device kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // load the answer back into the host
    err = hipMemcpy(h_dst, d_dst, nBytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector DST from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_src);
    hipFree(d_dst);
}
